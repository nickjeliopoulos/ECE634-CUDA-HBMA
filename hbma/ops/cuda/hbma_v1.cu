#include "hip/hip_runtime.h"

#include "hbma.cuh"
#include "cuda_helpers.cuh"

namespace ops::cuda::hbma::v1 {
	namespace {
		// Constant expressions
		constexpr int HBMA_MAX_LEVELS = 1;
		constexpr int INPUT_CHANNELS = 3;

		// Structure to hold problem size parameters.
		struct hbma_problem_size {
			int levels;
			int image_channels;
			int image_height;
			int image_width;
			int block_size_height[HBMA_MAX_LEVELS];
			int block_size_width[HBMA_MAX_LEVELS];
			int block_counts_height[HBMA_MAX_LEVELS];
			int block_counts_width[HBMA_MAX_LEVELS];
			int neighborhood_sizes[HBMA_MAX_LEVELS];
			int neighborhood_including_self_size[HBMA_MAX_LEVELS];
			bool is_valid;
		};
	}

	// Helper function to compute and store the problem size.
	hbma_problem_size get_hbma_problem_size(
		const torch::Tensor& anchor_frame, 
		const torch::Tensor& target_frame,
		const int levels,
		const int block_size_height,
		const int block_size_width,
		const int neighborhood_size
	) {
		// Tensors have NCHW layout
		const int C = (int) target_frame.size(1);
		const int H = (int) target_frame.size(2);
		const int W = (int) target_frame.size(3);

		const int block_count_height = H / block_size_height;
		const int block_count_width = W / block_size_width;

		bool valid_problem_size = (
			// Input Sizes
			H % block_size_height == 0 && 
			W % block_size_width == 0 &&
			block_count_height > 0 &&
			block_count_width > 0 &&
			// TODO: Remove restriction in future. Implement kernel emitting (or JIT), or just live with the fact we might not be able to unroll certain loops
			C == INPUT_CHANNELS 
		);

		const hbma_problem_size problem_size = {
			levels,
			C, H, W,
			{block_size_height}, {block_size_width},
			{block_count_height}, {block_count_width},
			{neighborhood_size},
			// For each pixel block, compute the costs of all blocks in the search window around it
			// This is the total number of neighbors around each block including itself
			// NOTE: You need to do bounds checking, because blocks near edges will have fewer neighbors
			{(2 * neighborhood_size + 1) * (2 * neighborhood_size + 1)},
			valid_problem_size
		};

		return problem_size;
	}

	// Step 1
	// This is really just a standard elementwise kernel, with some extra indexing flavor
	// We should probably look at the PyTorch elementwise kernels for a reference
	__global__ void _hbma_compute_block_cost_kernel(
		const torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> anchor_frame, 
		const torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> target_frame, 
		torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> neighborhood_block_costs,
		const int level,
		const int image_channels, 
		const int image_height, 
		const int image_width,
		const int block_size_height,
		const int block_size_width,
		const int block_count_height,
		const int block_count_width,
		const int neighborhood_size
	) {
		// Base block index (in block space)
		int block_idx_h = blockIdx.y;  // vertical block index
		int block_idx_w = blockIdx.x;  // horizontal block index
	
		// Neighborhood offset: derive from blockIdx.z
		int offset_idx_h = (blockIdx.z / (2 * neighborhood_size + 1)) - neighborhood_size;
		int offset_idx_w = (blockIdx.z % (2 * neighborhood_size + 1)) - neighborhood_size;
		int neighbor_block_idx_h = block_idx_h + offset_idx_h;
		int neighbor_block_idx_w = block_idx_w + offset_idx_w;
		
		// Check bounds for neighbor block
		if (neighbor_block_idx_h < 0 || neighbor_block_idx_h >= block_count_height ||
			neighbor_block_idx_w < 0 || neighbor_block_idx_w >= block_count_width) {
			return;
		}
	
		// Compute starting pixel coordinates for the anchor and neighbor blocks.
		int anchor_block_start_h = block_idx_h * block_size_height;
		int anchor_block_start_w = block_idx_w * block_size_width;
		int neighbor_block_start_h = neighbor_block_idx_h * block_size_height;
		int neighbor_block_start_w = neighbor_block_idx_w * block_size_width;
	
		// Global pixel coordinates for this thread within the block.
		int thread_anchor_pixel_idx_h = anchor_block_start_h + threadIdx.y;
		int thread_anchor_pixel_idx_w = anchor_block_start_w + threadIdx.x;
		int thread_neighbor_pixel_idx_h = neighbor_block_start_h + threadIdx.y;
		int thread_neighbor_pixel_idx_w = neighbor_block_start_w + threadIdx.x;
		
		// Threadblock shared memory for cost accumulation
		// NOTE: This may be smaller or bigger than the block size - configure this at some point if we need larger sizes
		__shared__ float cost_cache[32][32];
	
		// Compute per-thread cost (sum-squared differences) across channels
		float thread_cost = 0.0f;
		#pragma unroll
		for (int c = 0; c < INPUT_CHANNELS; c++) {
			float difference = target_frame[0][c][thread_neighbor_pixel_idx_h][thread_neighbor_pixel_idx_w] -
							   anchor_frame[0][c][thread_anchor_pixel_idx_h][thread_anchor_pixel_idx_w];
			thread_cost += difference * difference;
		}
	
		// Store the cost in shared memory and synchronize.
		cost_cache[threadIdx.y][threadIdx.x] = thread_cost;
		__syncthreads();
		
		// Thread (0,0) performs reduction over the block.
		if (threadIdx.x + threadIdx.y == 0) {
			float total_cost = 0.0f;
			for (int i = 0; i < block_size_height; i++) {
				for (int j = 0; j < block_size_width; j++) {
					total_cost += cost_cache[i][j];
				}
			}
			// Write the computed cost to the output tensor
			neighborhood_block_costs[0][blockIdx.y][blockIdx.x][blockIdx.z] = total_cost;
		}
	}
	
	// Step 3
	__global__ void _hbma_compute_reconstructed_frame_kernel(
		// Anchor frame: [N, C, H, W]
		const torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> anchor_frame, 
		// Best neighbor indices: [N, block_count_height, block_count_width]
		const torch::PackedTensorAccessor64<long long, 3, torch::RestrictPtrTraits> neighborhood_block_cost_indices,
		// Output reconstructed frame: [N, C, H, W]
		torch::PackedTensorAccessor64<float, 4, torch::RestrictPtrTraits> reconstructed_frame,
		const int level,
		const int image_channels, 
		const int image_height, 
		const int image_width,
		const int block_size_height,
		const int block_size_width,
		const int block_count_height,
		const int block_count_width,
		const int neighborhood_size
	) {
		// Each grid block corresponds to one block in the image.
		int block_idx_h = blockIdx.y;  // vertical block index
		int block_idx_w = blockIdx.x;  // horizontal block index
	
		// Compute the top-left pixel coordinate of the destination block in the reconstructed frame.
		int dest_block_start_h = block_idx_h * block_size_height;
		int dest_block_start_w = block_idx_w * block_size_width;
	
		// Read the best matching neighbor index for this block.
		// Our argmin tensor has shape [1, block_count_height, block_count_width].
		long best_neighbor_index = neighborhood_block_cost_indices[0][block_idx_h][block_idx_w];
		
		// Decode the best neighbor index into vertical and horizontal offsets.
		// The total number of neighbors is: (2*neighborhood_size+1)^2.
		int neighbor_dim = 2 * neighborhood_size + 1;
		int offset_idx_h = (best_neighbor_index / neighbor_dim) - neighborhood_size;
		int offset_idx_w = (best_neighbor_index % neighbor_dim) - neighborhood_size;
	
		// Compute the neighbor block indices from the current block indices.
		int neighbor_block_idx_h = block_idx_h + offset_idx_h;
		int neighbor_block_idx_w = block_idx_w + offset_idx_w;
	
		// Compute the starting pixel coordinates of the neighbor block in the anchor frame.
		int neighbor_block_start_h = neighbor_block_idx_h * block_size_height;
		int neighbor_block_start_w = neighbor_block_idx_w * block_size_width;
	
		// Each thread corresponds to one pixel within the block.
		int local_y = threadIdx.y;
		int local_x = threadIdx.x;
		int dest_y = dest_block_start_h + local_y;
		int dest_x = dest_block_start_w + local_x;
	
		// Check that the neighbor block is within valid bounds.
		if (neighbor_block_idx_h >= 0 || neighbor_block_idx_h < block_count_height ||
			neighbor_block_idx_w >= 0 || neighbor_block_idx_w < block_count_width) {
			// Compute the corresponding source pixel coordinates in the anchor frame.
			int source_y = neighbor_block_start_h + local_y;
			int source_x = neighbor_block_start_w + local_x;

			// Copy across all channels
			#pragma unroll
			for (int c = 0; c < INPUT_CHANNELS; c++) {
				reconstructed_frame[0][c][dest_y][dest_x] = anchor_frame[0][c][source_y][source_x];
			}
		}
	}

	// HBMA v1 CUDA kernel operator
	// Operates on 4D anchor and target frames with shape [N,C,H,W]
	// Restrictions: Only supports N=1, C=3, and a single level currently
	torch::Tensor hbma_v1(
		const torch::Tensor& anchor_frame, 
		const torch::Tensor& target_frame,
		const int levels,
		const int block_size_height,
		const int block_size_width,
		const int neighborhood_size
	) {
		// Compute problem size, check validity
		const hbma_problem_size problem_size = get_hbma_problem_size(
			anchor_frame, 
			target_frame, 
			levels, 
			block_size_height, 
			block_size_width,
			neighborhood_size
		);
		
		// Throw runtime error if problem size is invalid
		if (!problem_size.is_valid) {
			throw std::runtime_error("Invalid problem size: Ensure input dimensions and parameters meet the requirements.");
		}

		// Allocate output and intermediate Tensors
		torch::Tensor reconstructed_frame = torch::empty_like(target_frame);		
		torch::Tensor neighborhood_block_costs = torch::full(
			{
				(int) anchor_frame.size(0), 
				problem_size.block_counts_height[0], 
				problem_size.block_counts_width[0], 
				problem_size.neighborhood_including_self_size[0]
			},
			// Initialize values to 1e9f (or a large value) - this behavior is required for block cost computation
			1e9f, 
			// Make sure this new tensor is on the same device as the target frame, and has the right data type
			torch::TensorOptions().dtype(torch::kFloat32).device(target_frame.device())
		);

		// Kernel Launch Bounds
		dim3 neighborhood_cost_threads(
			problem_size.block_size_width[0], 
			problem_size.block_size_height[0]
		);

		dim3 neighborhood_cost_grid(
			problem_size.block_counts_width[0], 
			problem_size.block_counts_height[0], 
			problem_size.neighborhood_including_self_size[0]
		);

		dim3 reconstruct_threads(
			problem_size.block_size_width[0], 
			problem_size.block_size_height[0]
		);

		// One grid block per image block
		dim3 reconstruct_grid(
			problem_size.block_counts_width[0], 
			problem_size.block_counts_height[0]
		);

		// Step 1: Get block costs
		_hbma_compute_block_cost_kernel<<<neighborhood_cost_grid, neighborhood_cost_threads>>>(  
			anchor_frame.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
			target_frame.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
			neighborhood_block_costs.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
			problem_size.levels,
			problem_size.image_channels, problem_size.image_height, problem_size.image_width,
			problem_size.block_size_height[0], problem_size.block_size_width[0],
			problem_size.block_counts_height[0], problem_size.block_counts_width[0],
			problem_size.neighborhood_sizes[0]
		);
		CUDA_LAUNCH_AND_EXECUTE_CHECK();

		// Step 2: Compute the argmin along the last dimension dim=3 (neighborhood cost dimension, this will yield the best matching index)
		torch::Tensor lowest_cost_neighborhood_block_indices = std::get<1>(torch::min(neighborhood_block_costs, 3));

		// Step 3: Compute motion vectors
		_hbma_compute_reconstructed_frame_kernel<<<reconstruct_grid, reconstruct_threads>>>(
			anchor_frame.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
			lowest_cost_neighborhood_block_indices.packed_accessor64<long long, 3, torch::RestrictPtrTraits>(),
			reconstructed_frame.packed_accessor64<float, 4, torch::RestrictPtrTraits>(),
			problem_size.levels,
			problem_size.image_channels, problem_size.image_height, problem_size.image_width,
			problem_size.block_size_height[0], problem_size.block_size_width[0],
			problem_size.block_counts_height[0], problem_size.block_counts_width[0],
			problem_size.neighborhood_sizes[0]
		);
		CUDA_LAUNCH_AND_EXECUTE_CHECK();

		return reconstructed_frame;
	}
}
